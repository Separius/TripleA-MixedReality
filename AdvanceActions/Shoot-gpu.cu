#include "hip/hip_runtime.h"
#include <Shoot-gpu.h>

gpuShoot::gpuShoot(WorldData* newWorldData, BasicPlayer* newBasicPlayer)
{
    pWorldData = newWorldData;
    pBasicPlayer = newBasicPlayer;
    selectedPoint.x = 0;
    selectedPoint.y = 0;
    whichGoal = 'o'; //t stands for teamGoal and o stands for oppGoal
    callerUid = pWorldData->my_id();
    shootReachableRadius = 165;
    ballPos = pWorldData->basedCenter(pWorldData->ball());
}

gpuShoot::gpuShoot()
{
    selectedPoint.x = 0;
    selectedPoint.y = 0;
    whichGoal = 'o'; //t stands for teamGoal and o stands for oppGoal
    shootReachableRadius = 165;
}

Point gpuShoot::getSelectedPoint()
{
    return selectedPoint ;
}
void gpuShoot::goalChooser(char whichOne)
{
    whichGoal = whichOne ;
}

void gpuShoot::gpuKickNow(Point target)
{
    float angle = 0;

    Vector shootVector;

    shootVector.setByPoints(pWorldData->basedCenter(pWorldData->ball()), target);

    angle = pWorldData->bodyAngle() - shootVector.getDirection();
    pBasicPlayer->kick(angle * M_PI / 180, 1);
}

void gpuShoot::initializeData(WorldData* newWorldData, BasicPlayer* newBasicPlayer)
{
    pWorldData = newWorldData;
    callerUid = pWorldData->my_id();
    ballPos = pWorldData->basedCenter(pWorldData->ball());
    pBasicPlayer = newBasicPlayer;
}

void gpuShoot::setBallPos(Point newBallPos)
{
    ballPos = newBallPos ;
}

void gpuShoot::setReachableDist(float newReachable)
{
    shootReachableRadius = newReachable ;
}

void gpuShoot::setUid(int givenID)
{
    callerUid = givenID ;
}

float gpuShoot::shootLastValue()
{
    return bestValue ;
}

void gpuShoot::gpuShootNow()
{
    float angle = 0;
    
    Vector shootVector;

    pointSelector();

    shootVector.setByPoints(ballPos, selectedPoint);

    if (selectedPoint.getX() != 0)
    {
        angle = pWorldData->bodyAngle() - shootVector.getDirection();
        pBasicPlayer->kick(angle * M_PI / 180, 1);
    }
    else
    {
        pBasicPlayer->kick(M_PI / 2, 1);
        //we should pass the ball !
    }
}

__global__ void bestPointSelector(int* goalPolDist , int* oppNum , float* oppPosX , float* oppPosY , float* ballPosX , float* ballPosY , float* values , float* goalPos1X , float* goalPos1Y , float* goalPos2X , float* goalPos2Y)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int reachAble = 165 ;
    float temp , minValue = 99 ;
    float endY = *goalPos2Y + 15.0 + idx ;
    
    if (endY < (*goalPos1Y - 15.0))
    {
        Point start , end ;
        start.x = *ballPosX ;
        start.y = *ballPosY ;
        end.x = *goalPos1X ;
        end.y = endY ;
        Line shootLine;
        Circle startCircle, endCircle;
        Vector tempVector;
        Vector shootVector;	

        shootVector.x = end.x - start.x ;
        shootVector.y = end.y - start.y ;
        shootVector.magnitude = hypot(shootVector.x , shootVector.y) ;
        temp = atan2(shootVector.y , shootVector.x) * 180 / M_PI ;
        while (temp < -180) temp += 360 ;
        while (temp > 180) temp -= 360 ;
        shootVector.direction = temp ;

	if (end.x == start.x)
	{
		shootLine.b = 0;
		shootLine.a = 1;
		shootLine.c = 0 - start.x;
		shootLine.m = 0xFFFFFF;
	}
	else
	{
		shootLine.m = (end.y - start.y) / (end.x - start.x);
		shootLine.a = start.y - end.y;
		shootLine.b = end.x - start.x;
		shootLine.c = start.y * (start.x - end.x) + start.x * (end.y - start.y);
	}
	shootLine.direction = atan2(end.y - start.y , end.x - start.x) * 180 / M_PI ;

	tempVector.x = shootVector.x / 16.f ;
	tempVector.y = shootVector.y / 16.f ;
	tempVector.magnitude = hypot(tempVector.x, tempVector.y);
        temp = atan2(tempVector.y, tempVector.x) * 180 / M_PI ;
	while (temp < -180) temp += 360 ;
        while (temp > 180) temp -= 360 ;
        tempVector.direction = temp ;

	tempVector.x *= 5 ;
	tempVector.y *= 5 ;
	tempVector.magnitude = hypot(tempVector.x, tempVector.y);
	temp = atan2(tempVector.y, tempVector.x) * 180 / M_PI ;
	while (temp < -180) temp += 360 ;
        while (temp > 180) temp -= 360 ;
        tempVector.direction = temp ;
        
        Point startCenter ;
	startCenter.x = start.x + tempVector.x ;
	startCenter.y = start.y + tempVector.y ;

        Point endCenter = end;
        Point thisOppCenter;

        float d = hypot(start.x - end.x , start.y - end.y);
        
        if (d > reachAble)
            minValue = 0 ;
        else
        {
            float startRadius = 5 * d / 16;
            if (startRadius < 35)
                startRadius = 40 ;
            float endRadius = d / 2;
            if (endRadius < 35)
                endRadius = 40 ;
            float startZaribOpp = 1;
            float endZaribOpp = 0.5;

            startCircle.centeralPoint = startCenter ;
	    startCircle.radius = startRadius ;
	    endCircle.centeralPoint = endCenter ;
	    endCircle.radius = endRadius ;

            for (int i = 0; i < *oppNum; i++)
            {
                thisOppCenter.x = oppPosX[i];
                thisOppCenter.y = oppPosY[i];

                //34.5 = players max kickable area radius
                //this is completly correct if the opponents won't move !

		Line tempLine;
		Point interPoint;

		temp = shootLine.direction + 90 ;
		while (temp < -180) temp += 360 ;
		while (temp > 180) temp -= 360 ;
		
		float cs, sn, n;
		if (((temp + 90) / 180) == (int)((temp + 90) / 180))
		{
			tempLine.b = 0;
			tempLine.a = 1;
			tempLine.c = 0 - thisOppCenter.x;
			tempLine.m = 0xFFFFFF;
		}
		else
		{
			tempLine.m = tan(temp * M_PI/180);
			n = thisOppCenter.y - (tempLine.m * thisOppCenter.x);
			sn = sin(temp * M_PI/180);
			cs = cos(temp * M_PI/180);
			tempLine.a = 0 - sn;
			tempLine.b = cs;
			tempLine.c = 0 - (cs * n);
		}
		tempLine.direction = temp;
		
		float a1 = shootLine.a, b1 = shootLine.b, c1 = shootLine.c;
		float a2 = tempLine.a, b2 = tempLine.b, c2 = tempLine.c;
		interPoint.x = ((b1 * c2) - (b2 * c1)) / ((a1 * b2) - (a2 * b1));
		interPoint.y = ((a2 * c1) - (a1 * c2)) / ((a1 * b2) - (a2 * b1));

		float dist2Line = hypot(thisOppCenter.x - interPoint.x , thisOppCenter.y - interPoint.y) ;
                
		float centerToPointDist = sqrt((pow((startCircle.centeralPoint.x - thisOppCenter.x), 2) + pow((startCircle.centeralPoint.y - thisOppCenter.y), 2)));
		
                if (centerToPointDist < startCircle.radius)
                {
                    if (dist2Line < 40)
                    {
                        minValue = 0;
                        break;
                    }
                    if ((dist2Line * startZaribOpp) < minValue)
                    {
                        minValue = dist2Line * startZaribOpp + (1 - ( d / reachAble));
                    }
                }

                centerToPointDist = sqrt((pow((endCircle.centeralPoint.x - thisOppCenter.x), 2) + pow((endCircle.centeralPoint.y - thisOppCenter.y), 2)));
		if (centerToPointDist < endCircle.radius)
                {
                    if (dist2Line < 60)
                    {
                        minValue = 0;
                        break;
                    }
                    if ((dist2Line * endZaribOpp) < minValue)
                    {
                        minValue = dist2Line * endZaribOpp + (1 - ( d / reachAble));
                    }
                }
            }
            if (minValue == 99)
                minValue += (1 - ( d / reachAble)) ;
            
            values[idx] = minValue;
        }
    }
}

void gpuShoot::pointSelector()
{
    //Host Things :
    int maxValueId = 0 ;
    int *goalPolDist_h = (int *)malloc(sizeof(int)) ;
    int temp = pWorldData->mr_number_found_opponents();
    int* oppNum_h = &temp ;
    float* oppPosX_h ;
    float* oppPosY_h ;
    float* ballPosX_h = (float *)malloc(sizeof(float)) ;
    float* ballPosY_h = (float *)malloc(sizeof(float)) ;
    float* values_h ;
    float* goalPos1X_h = (float *)malloc(sizeof(float)) ;
    float* goalPos1Y_h = (float *)malloc(sizeof(float)) ;
    float* goalPos2X_h = (float *)malloc(sizeof(float)) ;
    float* goalPos2Y_h = (float *)malloc(sizeof(float)) ;

    *ballPosX_h = ballPos.getX();
    *ballPosY_h = ballPos.getY();
    if (whichGoal == 'o')
    {
        oppPosX_h = (float *)malloc(temp * sizeof(float)) ;
        oppPosY_h = (float *)malloc(temp * sizeof(float)) ;
        *goalPos1X_h = pWorldData->oppPole1.getX();
        *goalPos1Y_h = pWorldData->oppPole1.getY();
        *goalPos2X_h = pWorldData->oppPole2.getX();
        *goalPos2Y_h = pWorldData->oppPole2.getY();
    }
    if (whichGoal == 't')
    {
        temp = pWorldData->mr_number_found_teammates() + 1 ;
        oppPosX_h = (float *)malloc(temp * sizeof(float)) ;
        oppPosY_h = (float *)malloc(temp * sizeof(float)) ;
        *goalPos1X_h = pWorldData->basedCenter(pWorldData->mygoal_pole1()).getX();
        *goalPos1Y_h = pWorldData->basedCenter(pWorldData->mygoal_pole1()).getY();
        *goalPos2X_h = pWorldData->basedCenter(pWorldData->mygoal_pole2()).getX();
        *goalPos2Y_h = pWorldData->basedCenter(pWorldData->mygoal_pole2()).getY();
    }
    
    *goalPolDist_h = (int)(*goalPos1Y_h - *goalPos2Y_h) - 30 ;

    values_h = (float *)malloc( *goalPolDist_h * sizeof(float));

    if(whichGoal == 'o')
    {
        for (int i = 0 ; i < *oppNum_h ; i++)
        {
            oppPosX_h[i] = pWorldData->oppPositions[i].getX() ;
            oppPosY_h[i] = pWorldData->oppPositions[i].getY() ;
        }
    }
    if (whichGoal == 't')
    {
        for(int i = 0 ; i < pWorldData->mr_number_found_teammates() ; i++)
        {
            oppPosX_h[i] = pWorldData->tmmPositions[i].getX() ;
            oppPosY_h[i] = pWorldData->tmmPositions[i].getY() ;
        }
    }

    for (int i = 0 ; i < *goalPolDist_h ; i++)
        values_h[i] = 0 ;

    //Device Things :
    int* goalPolDist_d ;
    int* oppNum_d ;
    float* oppPosX_d ;
    float* oppPosY_d ;
    float* ballPosX_d;
    float* ballPosY_d;
    float* values_d;
    float* goalPos1X_d;
    float* goalPos1Y_d ;
    float* goalPos2X_d ;
    float* goalPos2Y_d ;

    hipMalloc((void**)&goalPolDist_d , sizeof(int));
    hipMalloc((void**)&oppNum_d , sizeof(int));
    hipMalloc((void**)&oppPosX_d , temp * sizeof(float));
    hipMalloc((void**)&oppPosY_d , temp * sizeof(float));
    hipMalloc((void**)&ballPosX_d , sizeof(float));
    hipMalloc((void**)&ballPosY_d , sizeof(float));
    hipMalloc((void**)&values_d , *goalPolDist_h * sizeof(float));
    hipMalloc((void**)&goalPos1X_d , sizeof(float));
    hipMalloc((void**)&goalPos1Y_d , sizeof(float));
    hipMalloc((void**)&goalPos2X_d , sizeof(float));
    hipMalloc((void**)&goalPos2Y_d , sizeof(float));
    

    //Copy From Host to Device :
    hipMemcpy(goalPolDist_d , goalPolDist_h , sizeof(int) , hipMemcpyHostToDevice);
    hipMemcpy(oppNum_d , oppNum_h , sizeof(int) , hipMemcpyHostToDevice);
    hipMemcpy(oppPosX_d , oppPosX_h , temp * sizeof(float) , hipMemcpyHostToDevice);
    hipMemcpy(oppPosY_d , oppPosY_h , temp * sizeof(float) , hipMemcpyHostToDevice);
    hipMemcpy(ballPosX_d , ballPosX_h , sizeof(float) , hipMemcpyHostToDevice);
    hipMemcpy(ballPosY_d , ballPosY_h , sizeof(float) , hipMemcpyHostToDevice);
    hipMemcpy(values_d , values_h , sizeof(float) * *goalPolDist_h , hipMemcpyHostToDevice);
    hipMemcpy(goalPos1X_d , goalPos1X_h , sizeof(float) , hipMemcpyHostToDevice);
    hipMemcpy(goalPos1Y_d , goalPos1Y_h , sizeof(float) , hipMemcpyHostToDevice);
    hipMemcpy(goalPos2X_d , goalPos2X_h , sizeof(float) , hipMemcpyHostToDevice);
    hipMemcpy(goalPos2Y_d , goalPos2Y_h , sizeof(float) , hipMemcpyHostToDevice);
    

    //Call the Kernel :
    dim3 dimGrid(1);
    dim3 dimBlock(192);
    bestPointSelector<<<dimGrid , dimBlock>>>(goalPolDist_d , oppNum_d , oppPosX_d , oppPosY_d , ballPosX_d , ballPosY_d , values_d , goalPos1X_d , goalPos1Y_d , goalPos2X_d , goalPos2Y_d);
    hipDeviceSynchronize();

    //Copy values back to Host :
    hipMemcpy(values_h , values_d ,*goalPolDist_h * sizeof(float) , hipMemcpyDeviceToHost);
    //Compare Values and set the Point :
    for(int i = 1 ; i < *goalPolDist_h ; i++)
    {
        if (values_h[i] > values_h[maxValueId])
        {
            maxValueId = i ;
            bestValue = values_h[i] ;
        }
    }
    if (values_h[maxValueId] == 0)
    {
        //can not shoot ;
        selectedPoint.x = 0 ;
	selectedPoint.y = 0 ;
    }
    else
    {
        selectedPoint.x = pWorldData->oppPole1.getX();
        selectedPoint.y = pWorldData->oppPole2.getY() + 15 + maxValueId ;
    }
    
    hipFree(goalPolDist_d);
    hipFree(oppNum_d);
    hipFree(oppPosX_d);
    hipFree(oppPosY_d);
    hipFree(ballPosX_d);
    hipFree(ballPosY_d);
    hipFree(values_d);
    hipFree(goalPos1X_d);
    hipFree(goalPos1Y_d);
    hipFree(goalPos2X_d);
    hipFree(goalPos2Y_d);

    free(goalPolDist_h);
    free(oppPosX_h);
    free(oppPosY_h);
    free(ballPosX_h);
    free(ballPosY_h);
    free(values_h);
    free(goalPos1X_h);
    free(goalPos1Y_h);
    free(goalPos2X_h);
    free(goalPos2Y_h);
}