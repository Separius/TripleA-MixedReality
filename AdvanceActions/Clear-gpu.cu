#include "hip/hip_runtime.h"
#include "Clear-gpu.h"

gpuClear::gpuClear(WorldData* newWorldData, BasicPlayer* newBasicPlayer)
{
    pWorldData = newWorldData;
    pBasicPlayer = newBasicPlayer;
    selectedPoint.x = 0;
    selectedPoint.y = 0;
    clearReachableRadius = 165;
    ballPos = pWorldData->basedCenter(pWorldData->ball());
}

gpuClear::gpuClear()
{
    selectedPoint.x = 0;
    selectedPoint.y = 0;
    clearReachableRadius = 165;
}

Point gpuClear::getSelectedPoint()
{
    return selectedPoint ;
}

void gpuClear::initializeData(WorldData* newWorldData, BasicPlayer* newBasicPlayer)
{
    pWorldData = newWorldData;
    ballPos = pWorldData->basedCenter(pWorldData->ball());
    pBasicPlayer = newBasicPlayer;
}

void gpuClear::setBallPos(Point newBallPos)
{
    ballPos = newBallPos ;
}

void gpuClear::setReachableDist(int newReachable)
{
    clearReachableRadius = newReachable ;
}

float gpuClear::clearLastValue()
{
    return bestValue ;
}

void gpuClear::gpuClearNow()
{
    float angle = 0;

    Vector clearVector;

    pointSelector();

    clearVector.setByPoints(ballPos, selectedPoint);

    angle = pWorldData->bodyAngle() - clearVector.getDirection();
    pBasicPlayer->kick(angle * M_PI / 180, 1);
}

__global__ void bestPointSelector(int* oppNum , float* oppPosX ,float* oppPosY ,float* ballPosX ,float* ballPosY ,float* values ,float* goalPosTmmX ,float* goalPosOppX ,float* topY ,float* downY ,int reachAble)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    float temp , minValue = 99 ;
    float xMyPoint , yMyPoint ;

    if (idx < reachAble * 4)
    {
        xMyPoint = int(idx / 2) - reachAble ;
        yMyPoint = sqrt(reachAble * reachAble - xMyPoint * xMyPoint);
        xMyPoint = *ballPosX + xMyPoint ;
        if (idx % 2 == 0)
            yMyPoint = *ballPosY + yMyPoint ;
        else
            yMyPoint = *ballPosY - yMyPoint ;

        Point start , end ;
        start.x = *ballPosX ;
        start.y = *ballPosY ;
	
        Line clearLine;
        Circle startCircle, endCircle;
        Vector tempVector;
        Vector clearVector;

        clearVector.x = xMyPoint - start.x ;
        clearVector.y = yMyPoint - start.y ;
        clearVector.magnitude = hypot(clearVector.x , clearVector.y) ;
        temp = atan2(clearVector.y , clearVector.x) * 180 / M_PI ;
        while (temp < -180) temp += 360 ;
        while (temp > 180) temp -= 360 ;
        clearVector.direction = temp ;

	if (xMyPoint == start.x)
	{
		clearLine.b = 0;
		clearLine.a = 1;
		clearLine.c = 0 - start.x;
		clearLine.m = 0xFFFFFF;
	}
	else
	{
		clearLine.m = (yMyPoint - start.y) / (xMyPoint - start.x);
		clearLine.a = start.y - yMyPoint;
		clearLine.b = xMyPoint - start.x;
		clearLine.c = start.y * (start.x - xMyPoint) + start.x * (yMyPoint - start.y);
	}
	clearLine.direction = atan2(yMyPoint - start.y , xMyPoint - start.x) * 180 / M_PI ;

	tempVector.x = clearVector.x / 16.f ;
	tempVector.y = clearVector.y / 16.f ;
	tempVector.magnitude = hypot(tempVector.x, tempVector.y);
        temp = atan2(tempVector.y, tempVector.x) * 180 / M_PI ;
	while (temp < -180) temp += 360 ;
        while (temp > 180) temp -= 360 ;
        tempVector.direction = temp ;

	tempVector.x *= 5 ;
	tempVector.y *= 5 ;
	tempVector.magnitude = hypot(tempVector.x, tempVector.y);
	temp = atan2(tempVector.y, tempVector.x) * 180 / M_PI ;
	while (temp < -180) temp += 360 ;
        while (temp > 180) temp -= 360 ;
        tempVector.direction = temp ;

        Point startCenter ;
	startCenter.x = start.x + tempVector.x ;
	startCenter.y = start.y + tempVector.y ;

        Point endCenter;
	endCenter.x = xMyPoint ;
	endCenter.y = yMyPoint ;
        Point thisOppCenter;

        float d = hypot(start.x - xMyPoint , start.y - yMyPoint);
	float dist2Goal = hypot(*goalPosTmmX - xMyPoint , 0 - yMyPoint);

        if (d > reachAble)
            minValue = 0 ;
        else
        {
            float startRadius = 5 * d / 16;
            if (startRadius < 35)
                startRadius = 40 ;
            float endRadius = d / 2;
            if (endRadius < 35)
                endRadius = 40 ;
            float startZaribOpp = 1;
            float endZaribOpp = 0.5;

            startCircle.centeralPoint = startCenter ;
	    startCircle.radius = startRadius ;
	    endCircle.centeralPoint = endCenter ;
	    endCircle.radius = endRadius ;

            for (int i = 0; i < *oppNum; i++)
            {
                thisOppCenter.x = oppPosX[i];
                thisOppCenter.y = oppPosY[i];

		Line tempLine;
		Point interPoint;

		temp = clearLine.direction + 90 ;
		while (temp < -180) temp += 360 ;
		while (temp > 180) temp -= 360 ;

		float cs, sn, n;
		if (((temp + 90) / 180) == (int)((temp + 90) / 180))
		{
			tempLine.b = 0;
			tempLine.a = 1;
			tempLine.c = 0 - thisOppCenter.x;
			tempLine.m = 0xFFFFFF;
		}
		else
		{
			tempLine.m = tan(temp * M_PI/180);
			n = thisOppCenter.y - (tempLine.m * thisOppCenter.x);
			sn = sin(temp * M_PI/180);
			cs = cos(temp * M_PI/180);
			tempLine.a = 0 - sn;
			tempLine.b = cs;
			tempLine.c = 0 - (cs * n);
		}
		tempLine.direction = temp;

		float a1 = clearLine.a, b1 = clearLine.b, c1 = clearLine.c;
		float a2 = tempLine.a, b2 = tempLine.b, c2 = tempLine.c;
		interPoint.x = ((b1 * c2) - (b2 * c1)) / ((a1 * b2) - (a2 * b1));
		interPoint.y = ((a2 * c1) - (a1 * c2)) / ((a1 * b2) - (a2 * b1));

		float dist2Line = hypot(thisOppCenter.x - interPoint.x , thisOppCenter.y - interPoint.y) ;

		float centerToPointDist = sqrt((pow((startCircle.centeralPoint.x - thisOppCenter.x), 2) + pow((startCircle.centeralPoint.y - thisOppCenter.y), 2)));

                if (centerToPointDist < startCircle.radius)
                {
                    if (dist2Line < 40)
                    {
                        minValue = 0;
                        break;
                    }
                    if ((dist2Line * startZaribOpp) < minValue)
                    {
                        minValue = dist2Line * startZaribOpp + (dist2Goal / 100);
                    }
                }

                centerToPointDist = sqrt((pow((endCircle.centeralPoint.x - thisOppCenter.x), 2) + pow((endCircle.centeralPoint.y - thisOppCenter.y), 2)));
		if (centerToPointDist < endCircle.radius)
                {
                    if (dist2Line < 60)
                    {
                        minValue = 0;
                        break;
                    }
                    if ((dist2Line * endZaribOpp) < minValue)
                    {
                        minValue = dist2Line * endZaribOpp + (dist2Goal / 100);
                    }
                }
            }
            
            if (yMyPoint >= *topY || yMyPoint <= *downY)
	    {
		temp = 80;
		if (temp < minValue)
		{
		    minValue = temp + (dist2Goal / 100) ;
		}
	    }
	    
	    if (*goalPosTmmX < 0) //we are in the left side
	    {
	        if (xMyPoint <= *goalPosTmmX)
	        {
	            if (hypot(*goalPosTmmX - xMyPoint , 0 - yMyPoint) < 100) //then it is a goal to ourteam :D
	            {
	                minValue = 0;
	            }
	            else
	            {
	                temp = 20;
	                if (temp < minValue)
	                {
	                    minValue = temp + (dist2Goal / 100) ;
	                }
	            }
        	}
		else if (xMyPoint >= *goalPosOppX)
		{
		    temp = 60;
		    if (temp < minValue)
		    {
			minValue = temp + (dist2Goal / 100) ;
		    }
		}
	    }
	    else if (*goalPosTmmX > 0) //we are in the right side
	    {
	        if (xMyPoint >= *goalPosTmmX)
	        {
		    if (hypot(*goalPosTmmX - xMyPoint , 0 - yMyPoint) < 100) //then it is a goal to ourteam :D
	            {
	                minValue = 0;
	            }
	            else
	            {
	                temp = 20;
	                if (temp < minValue)
	                {
	                    minValue = temp + (dist2Goal / 100) ;
	                }
	            }
        	}
        	else if (xMyPoint <= *goalPosOppX)
	        {
	            temp = 60;
	            if (temp < minValue)
	            {
	                minValue = temp + (dist2Goal / 100) ;
	            }
        	}
	    }            
            if (minValue == 99)
                minValue += (dist2Goal / 100) ;

            values[idx] = minValue;
        }
    }
}

void gpuClear::pointSelector()
{
    //Host Things :
    cudaPrintfInit();
    int maxValueId = 0 ;
    int temp = pWorldData->mr_number_found_opponents();
    int* oppNum_h = &temp ;
    float* oppPosX_h = (float *)malloc(temp * sizeof(float)) ;
    float* oppPosY_h = (float *)malloc(temp * sizeof(float)) ;
    float* ballPosX_h = (float *)malloc(sizeof(float)) ;
    float* ballPosY_h = (float *)malloc(sizeof(float)) ;
    float* values_h ;
    float* goalPosTmmX_h = (float *)malloc(sizeof(float)) ;
    float* goalPosOppX_h = (float *)malloc(sizeof(float)) ;
    float* topY_h = (float *)malloc(sizeof(float)) ;
    float* downY_h = (float *)malloc(sizeof(float)) ;

    *topY_h = pWorldData->realVectors[10].getY();
    *downY_h = pWorldData->realVectors[0].getY();
    *ballPosX_h = ballPos.getX();
    *ballPosY_h = ballPos.getY();

    *goalPosTmmX_h = pWorldData->basedCenter(pWorldData->mygoal_pole1()).getX();
    *goalPosOppX_h = pWorldData->oppPole1.getX();

    values_h = (float *)malloc( clearReachableRadius * 4 * sizeof(float));

    
    for (int i = 0 ; i < *oppNum_h ; i++)
    {
        oppPosX_h[i] = pWorldData->oppPositions[i].getX() ;
        oppPosY_h[i] = pWorldData->oppPositions[i].getY() ;
    }
    
    for (int i = 0 ; i < 4 * clearReachableRadius ; i++)
        values_h[i] = 0 ;

    //Device Things :
    int* oppNum_d ;
    float* oppPosX_d ;
    float* oppPosY_d ;
    float* ballPosX_d;
    float* ballPosY_d;
    float* values_d;
    float* goalPosTmmX_d;
    float* goalPosOppX_d ;
    float* topY_d ;
    float* downY_d ;

    hipMalloc((void**)&oppNum_d , sizeof(int));
    hipMalloc((void**)&oppPosX_d , temp * sizeof(float));
    hipMalloc((void**)&oppPosY_d , temp * sizeof(float));
    hipMalloc((void**)&ballPosX_d , sizeof(float));
    hipMalloc((void**)&ballPosY_d , sizeof(float));
    hipMalloc((void**)&values_d , clearReachableRadius * 4 * sizeof(float));
    hipMalloc((void**)&goalPosTmmX_d , sizeof(float));
    hipMalloc((void**)&goalPosOppX_d , sizeof(float));
    hipMalloc((void**)&topY_d , sizeof(float));
    hipMalloc((void**)&downY_d , sizeof(float));


    //Copy From Host to Device :
    hipMemcpy(oppNum_d , oppNum_h , sizeof(int) , hipMemcpyHostToDevice);
    hipMemcpy(oppPosX_d , oppPosX_h , temp * sizeof(float) , hipMemcpyHostToDevice);
    hipMemcpy(oppPosY_d , oppPosY_h , temp * sizeof(float) , hipMemcpyHostToDevice);
    hipMemcpy(ballPosX_d , ballPosX_h , sizeof(float) , hipMemcpyHostToDevice);
    hipMemcpy(ballPosY_d , ballPosY_h , sizeof(float) , hipMemcpyHostToDevice);
    hipMemcpy(values_d , values_h , sizeof(float) * 4 * clearReachableRadius , hipMemcpyHostToDevice);
    hipMemcpy(goalPosTmmX_d , goalPosTmmX_h , sizeof(float) , hipMemcpyHostToDevice);
    hipMemcpy(goalPosOppX_d , goalPosOppX_h , sizeof(float) , hipMemcpyHostToDevice);
    hipMemcpy(topY_d , topY_h , sizeof(float) , hipMemcpyHostToDevice);
    hipMemcpy(downY_d , downY_h , sizeof(float) , hipMemcpyHostToDevice);


    //Call the Kernel :
    dim3 dimGrid(4);
    dim3 dimBlock(192);
    bestPointSelector<<<dimGrid , dimBlock>>>(oppNum_d , oppPosX_d , oppPosY_d , ballPosX_d , ballPosY_d , values_d , goalPosTmmX_d , goalPosOppX_d , topY_d , downY_d , clearReachableRadius);
    hipDeviceSynchronize();

    //Copy values back to Host :
    hipMemcpy(values_h , values_d ,clearReachableRadius * 4 * sizeof(float) , hipMemcpyDeviceToHost);
    //Compare Values and set the Point :
    for(int i = 1 ; i < clearReachableRadius * 4 ; i++)
    {
        if (values_h[i] > values_h[maxValueId])
        {
            maxValueId = i ;
            bestValue = values_h[i] ;
        }
    }
    
    selectedPoint.x = int(maxValueId / 2) - clearReachableRadius ;
    selectedPoint.y = sqrt(clearReachableRadius * clearReachableRadius - selectedPoint.x * selectedPoint.x);
    selectedPoint.x += *ballPosX_h ;
    if (maxValueId % 2 == 0)
        selectedPoint.y += *ballPosY_h ;
    else
        selectedPoint.y = *ballPosY_h - selectedPoint.y ;

    hipFree(oppNum_d);
    hipFree(oppPosX_d);
    hipFree(oppPosY_d);
    hipFree(ballPosX_d);
    hipFree(ballPosY_d);
    hipFree(values_d);
    hipFree(goalPosTmmX_d);
    hipFree(goalPosOppX_d);
    hipFree(topY_d);
    hipFree(downY_d);

    free(oppPosX_h);
    free(oppPosY_h);
    free(ballPosX_h);
    free(ballPosY_h);
    free(values_h);
    free(goalPosTmmX_h);
    free(goalPosOppX_h);
    free(topY_h);
    free(downY_h);
}