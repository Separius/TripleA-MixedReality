#include "hip/hip_runtime.h"
#include <cuPrintf.cu>
#include "Pass-gpu.h"

gpuPass::gpuPass(WorldData* newWorldData, BasicPlayer* newBasicPlayer)
{
    pWorldData = newWorldData;
    pBasicPlayer = newBasicPlayer;
    callerId = pWorldData->my_id();
    passReachableRadius = 165;
    ballPos = pWorldData->basedCenter(pWorldData->ball());
    selectedId = -1 ;
}

gpuPass::gpuPass()
{
    selectedId = -1 ;
    passReachableRadius = 165;
}

void gpuPass::initializeData(WorldData* newWorldData, BasicPlayer* newBasicPlayer)
{
    pWorldData = newWorldData;
    pBasicPlayer = newBasicPlayer;
    callerId = pWorldData->my_id();
    ballPos = pWorldData->basedCenter(pWorldData->ball());
}

__global__ void bestTeammateFinder(float* oppPosX ,float* oppPosY ,int* oppNum ,int* caller ,float* ourPosX ,float* ourPosY ,int* ourNum ,float* ballX ,float* ballY ,int* ourIds ,float* ourGoalX ,float* values)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    Line AB ;
}

Point gpuPass::getSelectedPoint()
{
    return selectedPoint ;
}

int gpuPass::getSelectedId()
{
    return selectedId ;
}

void gpuPass::gpuPassNow()
{
    float angle = 0;

    Vector shootVector;

    bestTeammateSelector();

    shootVector.setByPoints(ballPos, selectedPoint);

    if (selectedId != -1)
    {
        angle = pWorldData->bodyAngle() - shootVector.getDirection();
        pBasicPlayer->kick(angle * M_PI / 180, 1);
    }
    else
    {
        pBasicPlayer->kick(M_PI / 2, 1);
        //we should call Clear
    }
}

void gpuPass::kickIt()
{
    float angle = 0;

    Vector shootVector;

    shootVector.setByPoints(ballPos, selectedPoint);

    angle = pWorldData->bodyAngle() - shootVector.getDirection();
    pBasicPlayer->kick(angle * M_PI / 180, 1);
}

float gpuPass::passLastValue()
{
    return bestValue ;
}

void gpuPass::setBallPos(Point newBallPos)
{
    ballPos = newBallPos ;
}

void gpuPass::setReachableDist(float newReachable)
{
    passReachableRadius = newReachable ;
}

void gpuPass::setUid(int newId)
{
    callerId = newId ;
}

void gpuPass::bestTeammateSelector()
{
    //Host define & init:
    std::vector<int> tmmIDs = pWorldData->mr_found_teammates() ;
    std::vector<int> oppIDs = pWorldData->mr_found_opponents() ;
    float* oppPosX_h ;
    float* oppPosY_h ;
    int* oppNum_h = (int *)malloc(sizeof(int));
    int* caller_h = (int *)malloc(sizeof(int)) ;
    //int* myId_h = (int *)malloc(sizeof(int));
    float* ourPosX_h ;
    float* ourPosY_h ;
    int* ourNum_h = (int *)malloc(sizeof(int)) ;
    float* ballX_h = (float *)malloc(sizeof(float)) ;
    float* ballY_h = (float *)malloc(sizeof(float)) ;
    int* ourIds_h ;
    float* ourGoalX_h = (float *)malloc(sizeof(float)) ;
    float* values_h = (float *)malloc(sizeof(float) * 5 * 128) ;
    int maxValueId = -1 ;
    
    *oppNum_h = oppIDs.size() ;
    *ourNum_h = tmmIDs.size() + 1 ;
    oppPosX_h = (float *)malloc(sizeof(float) * *oppNum_h);
    oppPosY_h = (float *)malloc(sizeof(float) * *oppNum_h);
    ourPosX_h = (float *)malloc(sizeof(float) * *ourNum_h);
    ourPosY_h = (float *)malloc(sizeof(float) * *ourNum_h);
    ourIds_h = (int *)malloc(sizeof(int) * *ourNum_h);

    for(int i = 0 ; i < tmmIDs.size() ; i++)
        ourIds_h[i] = tmmIDs[i];
    ourIds_h[ tmmIDs.size() ] = pWorldData->my_id();

    std::sort(ourIds_h , ourIds_h + *ourNum_h);

    for(int i = 0 ; i < *ourNum_h ; i++)
    {
        if(ourIds_h[i] == pWorldData->my_id())
        {
            ourPosX_h[i] = pWorldData->me(true).getX() ;
            ourPosY_h[i] = pWorldData->me(true).getY() ;
        }
        else
        {
            ourPosX_h[i] = pWorldData->basedCenter(pWorldData->teammate(ourIds_h[i])).getX();
            ourPosY_h[i] = pWorldData->basedCenter(pWorldData->teammate(ourIds_h[i])).getY();
        }
    }

    for(int i = 0 ; i < *oppNum_h ; i++)
    {
        oppPosX_h[i] = pWorldData->basedCenter(pWorldData->opponent(oppIDs[i])).getX();
        oppPosY_h[i] = pWorldData->basedCenter(pWorldData->opponent(oppIDs[i])).getY();
    }

    *ourGoalX_h = pWorldData->ourPole1.getX() ;

    for (int i = 0 ; i < 5 * 128 ; i++ )
        values_h[i] = 0 ;

    //*myId_h = pWorldData->my_id() ;
    *caller_h = callerId ;

    *ballX_h = ballPos.getX() ;
    *ballY_h = ballPos.getY() ;
    
    //Device define :
    cudaPrintfInit();
    float* oppPosX_d ;
    float* oppPosY_d ;
    int* oppNum_d ;
    int* caller_d ;
    //int* myId_d ;
    float* ourPosX_d ;
    float* ourPosY_d ;
    int* ourNum_d = (int *)malloc(sizeof(int)) ;
    float* ballX_d = (float *)malloc(sizeof(float)) ;
    float* ballY_d = (float *)malloc(sizeof(float)) ;
    int* ourIds_d = (int *)malloc(sizeof(int) * *ourNum_h) ;
    float* ourGoalX_d = (float *)malloc(sizeof(float)) ;
    float* values_d = (float *)malloc(sizeof(float) * 5 * 128) ;

    hipMalloc((void**)&oppPosX_d , sizeof(float) * *oppNum_h);
    hipMalloc((void**)&oppPosY_d , sizeof(float) * *oppNum_h);
    hipMalloc((void**)&oppNum_d , sizeof(int));
    hipMalloc((void**)&caller_d , sizeof(int));
    //hipMalloc((void**)&myId_d , sizeof(int));
    hipMalloc((void**)&ourPosX_d , sizeof(float) * *ourNum_h);
    hipMalloc((void**)&ourPosY_d , sizeof(float) * *ourNum_h);
    hipMalloc((void**)&ourNum_d , sizeof(int));
    hipMalloc((void**)&ballX_d , sizeof(float));
    hipMalloc((void**)&ballY_d , sizeof(float));
    hipMalloc((void**)&ourIds_d , sizeof(int) * *ourNum_h);
    hipMalloc((void**)&ourGoalX_d , sizeof(float));
    hipMalloc((void**)&values_d , sizeof(float) * 5 * 128);
    
    //HostToDevice :
    hipMemcpy(oppPosX_d , oppPosX_h , sizeof(float) * *oppNum_h, hipMemcpyHostToDevice);
    hipMemcpy(oppPosY_d , oppPosY_h , sizeof(float) * *oppNum_h, hipMemcpyHostToDevice);
    hipMemcpy(oppNum_d , oppNum_h , sizeof(int) , hipMemcpyHostToDevice);
    hipMemcpy(caller_d , caller_h , sizeof(int) , hipMemcpyHostToDevice);
    //hipMemcpy(myId_d , myId_h , sizeof(int) , hipMemcpyHostToDevice);
    hipMemcpy(ourPosX_d , ourPosX_h , sizeof(float) * *ourNum_h , hipMemcpyHostToDevice);
    hipMemcpy(ourPosY_d , ourPosY_h , sizeof(float) * *ourNum_h , hipMemcpyHostToDevice);
    hipMemcpy(ourNum_d , ourNum_h , sizeof(int) , hipMemcpyHostToDevice);
    hipMemcpy(ballX_d , ballX_h , sizeof(float) , hipMemcpyHostToDevice);
    hipMemcpy(ballY_d , ballY_h , sizeof(float) , hipMemcpyHostToDevice);
    hipMemcpy(ourIds_d , ourIds_h , sizeof(int) *  *ourNum_h, hipMemcpyHostToDevice);
    hipMemcpy(ourGoalX_d , ourGoalX_h , sizeof(float) , hipMemcpyHostToDevice);
    hipMemcpy(values_d , values_h , sizeof(float) * 5 * 128 , hipMemcpyHostToDevice);
    
    //Call the Kernel :
    dim3 dimGrid(5);
    dim3 dimBlock(128);
    bestTeammateFinder<<<dimGrid , dimBlock>>>(oppPosX_d , oppPosY_d , oppNum_d , caller_d , ourPosX_d , ourPosY_d , ourNum_d , ballX_d , ballY_d , ourIds_d , ourGoalX_d , values_d);
    hipDeviceSynchronize();
    //DeviceToHost :
    hipMemcpy(values_h , values_d , sizeof(float) * 5 * 128 , hipMemcpyDeviceToHost);
    //CompareValues :

                                    //not done yet

    //CleanUp :
    hipFree(oppPosX_d);
    hipFree(oppPosY_d);
    hipFree(oppNum_d);
    hipFree(caller_d);
    //hipFree(myId_d);
    hipFree(ourPosX_d);
    hipFree(ourPosY_d);
    hipFree(ourNum_d);
    hipFree(ballX_d);
    hipFree(ballY_d);
    hipFree(ourIds_d);
    hipFree(ourGoalX_d);
    hipFree(values_d);

    free(oppPosX_h);
    free(oppPosY_h);
    free(oppNum_h);
    free(caller_h);
    //free(myId_h);
    free(ourPosX_h);
    free(ourPosY_h);
    free(ourNum_h);
    free(ballX_h);
    free(ballY_h);
    free(ourIds_h);
    free(ourGoalX_h);
    free(values_h);
    
    cudaPrintfDisplay(stdout, true);
    cudaPrintfEnd();
}